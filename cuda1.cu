#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include "lodepng.h"
using namespace std;

__global__
void filtr(const unsigned char* dev_input, unsigned char* dev_output, int width, int height, int cern) {
    //Индекс треда внутри текущего блока
    const unsigned int linearX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int linearY = blockIdx.y * blockDim.y + threadIdx.y;
    if (linearX >= width || linearY >= height) {
        return;
    }
    if(cern == 1){
        if(linearY > 1 and linearY < (height - 2) and linearX > 1 and linearX < (width - 2)) {
            dev_output[3 * (linearY * width + linearX)] = 
                    (dev_input[3 * (linearY * width + linearX )] *(36) +
                    dev_input[3 * (linearY * width + linearX - 1)] * 24 +
                    dev_input[3 * ((linearY + 1) * width + linearX)] * 24 +
                    dev_input[3 * ((linearY - 1) * width + linearX)] * 24 +
                    dev_input[3 * (linearY * width + linearX + 1)] * 24 +

                    dev_input[3 * ((linearY - 1) * width + linearX - 1)] * 16 +
                    dev_input[3 * ((linearY + 1) * width + linearX - 1)] * 16 +
                    dev_input[3 * ((linearY - 1) * width + linearX + 1)] * 16 +
                    dev_input[3 * ((linearY + 1) * width + linearX + 1)] * 16 +

                    dev_input[3 * (linearY * width + linearX - 2)] * 6 +
                    dev_input[3 * ((linearY + 2) * width + linearX)] * 6 +
                    dev_input[3 * ((linearY - 2) * width + linearX)] * 6 +
                    dev_input[3 * (linearY * width + linearX + 2)] * 6 +


                    dev_input[3 * ((linearY + 2) * width + linearX - 2)] +
                    dev_input[3 * ((linearY + 2) * width + linearX + 2)] +
                    dev_input[3 * ((linearY - 2) * width + linearX - 2)] +
                    dev_input[3 * ((linearY - 2) * width + linearX + 2)] +


                    dev_input[3 * ((linearY + 2) * width + linearX - 1)] +
                    dev_input[3 * ((linearY + 2) * width + linearX + 1)] +
                    dev_input[3 * ((linearY - 2) * width + linearX - 1)] +
                    dev_input[3 * ((linearY - 2) * width + linearX + 1)] +

                    dev_input[3 * ((linearY + 1) * width + linearX - 2)] +
                    dev_input[3 * ((linearY + 1) * width + linearX + 2)] +
                    dev_input[3 * ((linearY - 1) * width + linearX - 2)] +
                    dev_input[3 * ((linearY - 1) * width + linearX + 2)] ) / 256;




             dev_output[3 * (linearY * width + linearX) + 1] = (
                    dev_input[3 * (linearY * width + linearX ) + 1] * 36 +
                    dev_input[3 * (linearY * width + linearX - 1) + 1] * 24 +
                    dev_input[3 * ((linearY + 1) * width + linearX) + 1] * 24 +
                    dev_input[3 * ((linearY - 1) * width + linearX) + 1] * 24 +
                    dev_input[3 * (linearY * width + linearX + 1) + 1] * 24 +

                    dev_input[3 * ((linearY - 1) * width + linearX - 1) + 1] * 16 +
                    dev_input[3 * ((linearY + 1) * width + linearX - 1) + 1] * 16 +
                    dev_input[3 * ((linearY - 1) * width + linearX + 1) + 1] * 16 +
                    dev_input[3 * ((linearY + 1) * width + linearX + 1) + 1] * 16 +

                    dev_input[3 * (linearY * width + linearX - 2) + 1] * 6 +
                    dev_input[3 * ((linearY + 2) * width + linearX) + 1] * 6 +
                    dev_input[3 * ((linearY - 2) * width + linearX) + 1] * 6 +
                    dev_input[3 * (linearY * width + linearX + 2) + 1] * 6 +


                    dev_input[3 * ((linearY + 2) * width + linearX - 2) + 1] +
                    dev_input[3 * ((linearY + 2) * width + linearX + 2) + 1] +
                    dev_input[3 * ((linearY - 2) * width + linearX - 2) + 1] +
                    dev_input[3 * ((linearY - 2) * width + linearX + 2) + 1] +


                    dev_input[3 * ((linearY + 2) * width + linearX - 1) + 1] * 4 +
                    dev_input[3 * ((linearY + 2) * width + linearX + 1) + 1] * 4 +
                    dev_input[3 * ((linearY - 2) * width + linearX - 1) + 1] * 4 +
                    dev_input[3 * ((linearY - 2) * width + linearX + 1) + 1] * 4 +

                    dev_input[3 * ((linearY + 1) * width + linearX - 2) + 1] * 4 +
                    dev_input[3 * ((linearY + 1) * width + linearX + 2) + 1] * 4 +
                    dev_input[3 * ((linearY - 1) * width + linearX - 2) + 1] * 4 +
                    dev_input[3 * ((linearY - 1) * width + linearX + 2) + 1] * 4) / 256;




            dev_output[3 * (linearY * width + linearX) + 2] = (
                    dev_input[3 * (linearY * width + linearX ) + 2] *(36) +
                    dev_input[3 * (linearY * width + linearX - 1) + 2] * 24 +
                    dev_input[3 * ((linearY + 1) * width + linearX) + 2] * 24 +
                    dev_input[3 * ((linearY - 1) * width + linearX) + 2] * 24 +
                    dev_input[3 * (linearY * width + linearX + 1) + 2] * 24 +

                    dev_input[3 * ((linearY - 1) * width + linearX - 1) + 2] * 16 +
                    dev_input[3 * ((linearY + 1) * width + linearX - 1) + 2] * 16 +
                    dev_input[3 * ((linearY - 1) * width + linearX + 1) + 2] * 16 +
                    dev_input[3 * ((linearY + 1) * width + linearX + 1) + 1] * 16 +

                    dev_input[3 * (linearY * width + linearX - 2) + 2] * 6 +
                    dev_input[3 * ((linearY + 2) * width + linearX) + 2] * 6 +
                    dev_input[3 * ((linearY - 2) * width + linearX) + 2] * 6 +
                    dev_input[3 * (linearY * width + linearX + 2) + 2] * 6 +


                    dev_input[3 * ((linearY + 2) * width + linearX - 2) + 2] +
                    dev_input[3 * ((linearY + 2) * width + linearX + 2) + 2] +
                    dev_input[3 * ((linearY - 2) * width + linearX - 2) + 2] +
                    dev_input[3 * ((linearY - 2) * width + linearX + 2) + 2] +


                    dev_input[3 * ((linearY + 2) * width + linearX - 1) + 2] * 4 +
                    dev_input[3 * ((linearY + 2) * width + linearX + 1) + 2] * 4 +
                    dev_input[3 * ((linearY - 2) * width + linearX - 1) + 2] * 4 +
                    dev_input[3 * ((linearY - 2) * width + linearX + 1) + 2] * 4 +

                    dev_input[3 * ((linearY + 1) * width + linearX - 2) + 2] * 4 +
                    dev_input[3 * ((linearY + 1) * width + linearX + 2) + 2] * 4 +
                    dev_input[3 * ((linearY - 1) * width + linearX - 2) + 2] * 4 +
                    dev_input[3 * ((linearY - 1) * width + linearX + 2) + 2] * 4 ) / 256;
        }
    }
    else if(cern == 2){
        if(linearY > 0 and linearY < (height - 1) and linearX > 0 and linearX < (width - 1)) {
            dev_output[3 * (linearY * width + linearX)] =
                    (dev_input[3 * (linearY * width + linearX)] +
                          dev_input[3 * (linearY * width + linearX + 1)] +
                          dev_input[3 * (linearY * width + linearX - 1)] +
                          dev_input[3 * ((linearY + 1) * width + linearX)] +
                          dev_input[3 * ((linearY - 1) * width + linearX)] +
                          dev_input[3 * ((linearY + 1) * width + linearX - 1)] +
                          dev_input[3 * ((linearY - 1) * width + linearX - 1)] +
                          dev_input[3 * ((linearY + 1) * width + linearX + 1)] +
                          dev_input[3 * ((linearY - 1) * width + linearX + 1)]) / 9;
            dev_output[3 * (linearY * width + linearX) + 1] =
                    (dev_input[3 * (linearY * width + linearX) + 1] +
                          dev_input[3 * (linearY * width + linearX + 1) + 1] +
                          dev_input[3 * (linearY * width + linearX - 1) + 1] +
                          dev_input[3 * ((linearY + 1) * width + linearX) + 1] +
                          dev_input[3 * ((linearY - 1) * width + linearX) + 1] +
                          dev_input[3 * ((linearY + 1) * width + linearX - 1) + 1] +
                          dev_input[3 * ((linearY - 1) * width + linearX - 1) + 1] +
                          dev_input[3 * ((linearY + 1) * width + linearX + 1) + 1] +
                          dev_input[3 * ((linearY - 1) * width + linearX + 1) + 1]) / 9;
            dev_output[3 * (linearY * width + linearX) + 2] =
                    (dev_input[3 * (linearY * width + linearX) + 2] +
                          dev_input[3 * (linearY * width + linearX + 1) + 2] +
                          dev_input[3 * (linearY * width + linearX - 1) + 2] +
                          dev_input[3 * ((linearY + 1) * width + linearX) + 2] +
                          dev_input[3 * ((linearY - 1) * width + linearX) + 2] +
                          dev_input[3 * ((linearY + 1) * width + linearX - 1) + 2] +
                          dev_input[3 * ((linearY - 1) * width + linearX - 1) + 2] +
                          dev_input[3 * ((linearY + 1) * width + linearX + 1) + 2] +
                          dev_input[3 * ((linearY - 1) * width + linearX + 1) + 2]) / 9;
        }
    }
    else if(cern == 3){
        if(linearY > 0 and linearY < (height - 1) and linearX > 0 and linearX < (width - 1)) {
            dev_output[3 * (linearY * width + linearX)] =
                    dev_input[3 * (linearY * width + linearX)] * 8 -
                    dev_input[3 * (linearY * width + linearX + 1)] -
                    dev_input[3 * (linearY * width + linearX - 1)] -
                    dev_input[3 * ((linearY + 1) * width + linearX)] -
                    dev_input[3 * ((linearY - 1) * width + linearX)] -
                    dev_input[3 * ((linearY + 1) * width + linearX - 1)] -
                    dev_input[3 * ((linearY - 1) * width + linearX - 1)] -
                    dev_input[3 * ((linearY + 1) * width + linearX + 1)] -
                    dev_input[3 * ((linearY - 1) * width + linearX + 1)];
            dev_output[3 * (linearY * width + linearX) + 1] =
                    dev_input[3 * (linearY * width + linearX) + 1] * 8 -
                    dev_input[3 * (linearY * width + linearX + 1) + 1] -
                    dev_input[3 * (linearY * width + linearX - 1) + 1] -
                    dev_input[3 * ((linearY + 1) * width + linearX) + 1] -
                    dev_input[3 * ((linearY - 1) * width + linearX) + 1] -
                    dev_input[3 * ((linearY + 1) * width + linearX - 1) + 1] -
                    dev_input[3 * ((linearY - 1) * width + linearX - 1) + 1] -
                    dev_input[3 * ((linearY + 1) * width + linearX + 1) + 1] -
                    dev_input[3 * ((linearY - 1) * width + linearX + 1) + 1];
            dev_output[3 * (linearY * width + linearX) + 2] =
                    dev_input[3 * (linearY * width + linearX) + 2] * 8 -
                    dev_input[3 * (linearY * width + linearX + 1) + 2] -
                    dev_input[3 * (linearY * width + linearX - 1) + 2] -
                    dev_input[3 * ((linearY + 1) * width + linearX) + 2] -
                    dev_input[3 * ((linearY - 1) * width + linearX) + 2] -
                    dev_input[3 * ((linearY + 1) * width + linearX - 1) + 2] -
                    dev_input[3 * ((linearY - 1) * width + linearX - 1) + 2] -
                    dev_input[3 * ((linearY + 1) * width + linearX + 1) + 2] -
                    dev_input[3 * ((linearY - 1) * width + linearX + 1) + 2];
        }
    }
}






void funk(const char* input_file, const char* output_file, int filtr_n){
    vector<unsigned char> in_image;
    unsigned int width, height;

    // Load the data
    unsigned error = lodepng::decode(in_image, width, height, input_file);
    float all_all = 0;
    float caunting_all = 0;
    hipEvent_t all_start;
    hipEvent_t all_stop;
    hipEvent_t start_caunting;
    hipEvent_t stop_caunting;
    hipEventCreate(&all_start);
    hipEventCreate(&all_stop);
    hipEventCreate(&start_caunting);
    hipEventCreate(&stop_caunting);


    // Prepare the data
    unsigned char* input_image = new unsigned char[(in_image.size()*3)/4];
    unsigned char* output_image = new unsigned char[(in_image.size()*3)/4];
    int where = 0;
    for(int i = 0; i < in_image.size(); ++i) {
        if((i+1) % 4 != 0) {
            input_image[where] = in_image.at(i);
            output_image[where] = 255;
            where++;
        }
    }

    hipEventRecord(all_start);
    hipEventSynchronize(all_start);


    unsigned char* dev_input;
    unsigned char* dev_output;
    hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char));
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int help = floor(sqrt(prop.maxThreadsPerBlock));
    if(help > height){
        help = height;
    }
    if(help > width){
        help = width;
    }  
    dim3 blockDims(help,help,1);
    dim3 gridDims(ceil(width / help), ceil(height / help), 1 );





    hipEventRecord(start_caunting);
    hipEventSynchronize(start_caunting);

    filtr<<<gridDims, blockDims>>>(dev_input, dev_output, width, height, filtr_n);
    hipEventRecord(stop_caunting);
    hipEventSynchronize(stop_caunting);
    hipEventElapsedTime(&caunting_all, start_caunting, stop_caunting);

    hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost );
    hipFree(dev_input);
    hipFree(dev_output);

    hipEventRecord(all_stop);
    hipEventSynchronize(all_stop);
    hipEventElapsedTime(&all_all, all_start, all_stop);


    vector<unsigned char> out_image;
    for(int i = 0; i < in_image.size(); ++i) {
        out_image.push_back(output_image[i]);
        if((i+1) % 3 == 0) {
            out_image.push_back(255);
        }
    }

    error = lodepng::encode(output_file, out_image, width, height);


    cout <<"caunting: " <<caunting_all << endl;
    cout << "all time "<< all_all << endl;

    delete[] input_image;
    delete[] output_image;

}








int main(int argc, char** argv) {
    string cern = argv[1];
    string size = argv[2];
    const char* input_file;
    const char* output_file;
    // Read the arguments
    int filtr_n = 0;
    if(cern == "blur5"){
        filtr_n = 1;
    }
    else if(cern == "blur"){
        filtr_n = 2;
    }
    else if(cern == "edge_detection"){
        filtr_n = 3;
    }
    if(size == "small"){
        input_file = "1.png";
        output_file = "1_out.png";
        funk(input_file, output_file, filtr_n);
        input_file = "1.png";
        output_file = "2_out.png";
        funk(input_file, output_file, filtr_n);
        input_file = "3.png";
        output_file = "3_out.png";
        funk(input_file, output_file, filtr_n);
    }
    else{
        input_file = "big.png";
        output_file = "big_out.png";
        funk(input_file, output_file, filtr_n);
    }
    
    return 0;
}
